#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#define N 100
__global__ void dkernel(int *da) {
  da[threadIdx.x] = threadIdx.x * threadIdx.x;
}

int main() {
  int a[N], *da;
  hipMalloc(&da, sizeof(int)*N);
  dkernel<<<1,N>>>(da);
  hipMemcpy(a,da,sizeof(int)*N,hipMemcpyDeviceToHost);
  for (int i = 0; i< N; i++) {
    std::cout << a[i] << std::endl;
  }
  return 0;
}

